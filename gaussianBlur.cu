#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <hip/hip_runtime.h>
#include <typeinfo>

using namespace std;
using namespace cv;


//Functions to print out char arrays for testing
void disp(char* data, int cols, int rows)
{
	for ( int i = 0 ; i < rows; i++)
	{
		for (int j = 0 ; j < cols; j++)
		{
			cout << (int)data[i*cols + j] << " ";
		}
		cout << endl;
	}
}

void disp(float* data, int cols, int rows)
{
	for ( int i = 0 ; i < rows; i++)
	{
		for (int j = 0 ; j < cols; j++)
		{
			cout << data[i*cols + j] << " ";
		}
		cout << endl;
	}
}

//Use OpenCV Mat class to save char array to file as greyscale image
void saveImage(string name, char* A, int rows, int cols)
{
	std::cout << "Printing to " << name << endl;
	cv::Mat greyImage (rows, cols, cv::DataType<uchar>::type);
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			greyImage.at<uchar>(i, j) = min(A[i*cols + j]*2, 255);
		}
	}
	cv::imwrite(name, greyImage);
}

//Convert data from OpenCV Mat class to char array
void readImage(Mat image, char* A, int rows, int cols)
{
	std::cout << "Initializing array" << endl;
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			A[i*cols + j] = image.at<uchar>(i, j)/2;
		}
	}
}

// The convolution matrix is identical, so we generate it once and pass through
// cudamemcpy to each vertex
void gaussianMatrix2d ( float* array, int radius, float sigSquare)
{
	int diameter = radius*2 + 1;
	float normalizationFactor = 0.0; // To make up for not convolving the whole matrix
	for (int i = -1* radius; i <= radius ; i ++)
	{
		for (int j = -1* radius ; j <= radius; j++)
		{
			array[(i + radius)* diameter + j + radius] = exp(-1.0*(i*i + j*j) / sigSquare);
			normalizationFactor += array[(i + radius)* diameter + j + radius];
		}
	}
	//Normalize, since the Gaussian is truncated, and we would like to integrate to 1
	for (int i = -1* radius; i <= radius ; i ++)
	{
		for (int j = -1* radius ; j <= radius; j++)
		{
			array[(i + radius)* diameter + j + radius] /= normalizationFactor;
		}
	}
}


//Main convolution function
__global__ void convolve2d(char* input, char* output, float* convolving, int width, int height, int radius, int diameter )
{
	//Which block are we in
	int blockX = blockIdx.x;
	int blockY = blockIdx.y;

	//which thread, of all threads, is this
	int threadX = blockX * blockDim.x + threadIdx.x;
	int threadY = blockY * blockDim.y + threadIdx.y;

	// How big an area does each thread have to cover?
	int sizeOfXSection = width  / (gridDim.x * blockDim.x); 
	int sizeOfYSection = height / (gridDim.y * blockDim.y);

	// if threadX < leftoverX, it takes on an extra thread
	int leftoverX = width  % (gridDim.x * blockDim.x); 
	int leftoverY = height % (gridDim.y * blockDim.y); 

	int startX = ( threadX) * sizeOfXSection + min (leftoverX, threadX );
	int startY = ( threadY) * sizeOfYSection + min (leftoverY, threadY );
	
	// if threadX < leftoverX, this thread should do an extra element
	int endX = startX + sizeOfXSection + (leftoverX > threadX );
	int endY = startY + sizeOfYSection + (leftoverY > threadY );

	float convolved = 0.f; // temp variable for result
	int locX, locY; // Local x and y
	
	// Main loop. Iterates over all pixels in its domain, and then convolves th
	// Gaussian matrix with the submatrix around the pixel.
	for (int curY = startY; curY < endY; curY++)
	{
		for (int curX = startX; curX < endX ; curX ++)
		{
			// Restart summing the convolved, iterate over all elements of submatrix
			convolved = 0.f;
			for (int ofsetY = -1* radius ; ofsetY <= radius; ofsetY++)
			{
				for (int ofsetX = -1* radius ; ofsetX <= radius ; ofsetX++)
				{
					locX = min ( width-1, max(0, curX + ofsetX));
					locY = min ( height-1,max(0, curY + ofsetY));
					convolved += input[locY* width + locX] * convolving[(ofsetY+radius)*diameter + radius + ofsetX];
				}
			}
			output[curY*width + curX] = (int)(convolved);
		}
	}
	return;

}

int main (int argc, char** argv)
{
	//parameters 
	int blurRadius = 5;
	float sigma = 1.0;
	string filename;
	if ( argc < 2){
		cout << "Please specify image file to work with"<<endl;
		return -1;
	}
	else {
		filename = argv[1];
	}
	if (argc > 2){
		blurRadius = atoi(argv[2]);
	}
	if (argc > 3){
		sigma = atoi(argv[3]);
	}

	//load image data to OpenCV matrix
	Mat image;
	image = imread(filename, 0);
	
	int imgWidth = image.cols;
	int imgHeight =image.rows;

	//Initialize variables and cuda arrays
	float sigSquare = 2 * sigma * sigma;
	int blurDiameter = 2* blurRadius + 1;
	int blurSize = blurDiameter* blurDiameter;
	int imgSize = imgWidth * imgHeight;

	char input[imgSize], output[imgSize];
	float gaussian[blurSize];
	char* deviceInput, *deviceOutput;
	float* deviceGaussian;

	hipMalloc( (void**) &deviceInput, imgSize * sizeof(char) );
	hipMalloc( (void**) &deviceOutput, imgSize * sizeof(char) );
	hipMalloc( (void**) &deviceGaussian, blurSize *sizeof(float) );

	// Copy from OpenCV matrix to regular old char array
	readImage(image, input, imgHeight, imgWidth);

	//Start timing
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventRecord(start,0);

	//Generate Gaussian matrix
	gaussianMatrix2d(gaussian, blurRadius, sigSquare);

	//Copy input and gaussian to device. Device does not need all of the input matrix
	// but some testing shows that copying the matrix accounts for less than 1%
	// of runtime
	hipMemcpy(deviceInput, input, imgSize*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(deviceGaussian, gaussian, blurSize*sizeof(float), hipMemcpyHostToDevice);
	// Mostly arbitrary numbers.
	dim3 threads(16, 16); 
	dim3 grid(8, 8);

	//Actually do the convolution
	convolve2d <<<grid, threads>>>(deviceInput, deviceOutput, deviceGaussian, imgWidth, imgHeight, blurRadius, blurDiameter);
	
	//Wait for all threads to end, then push output back to Host
	hipDeviceSynchronize(); 
	hipMemcpy(output, deviceOutput, imgSize*sizeof(char), hipMemcpyDeviceToHost);
	
	//Stop timing and report
	hipEventCreate(&stop);
 	hipEventRecord(stop,0);
 	hipEventSynchronize(stop);
 	hipEventElapsedTime(&elapsedTime, start,stop);
 	printf("Elapsed time : %f ms\n" ,elapsedTime);
	
	//Export images 
	saveImage("input.png", input, imgHeight, imgWidth);
	saveImage("output.png", output, imgHeight, imgWidth);
	return 1;
}